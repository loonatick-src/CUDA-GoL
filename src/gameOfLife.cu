#include "hip/hip_runtime.h"
#include "gameOfLife.h"


void
gameOfLifeBaseline(const int board_width,
        const int number_of_generations,
        const BoundaryCondition boundary_condition,
        Board_t initial_conditions)
{
    const dim3 dimBlock(32,32);
    const int gridWidth = ceil( ((float)board_width)/32 );
    const dim3 dimGrid(gridWidth, gridWidth);
     
    
}

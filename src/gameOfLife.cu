#include "hip/hip_runtime.h"
#include "gameOfLife.h"


void
gameOfLifeBaseline(const int board_width,
        const int number_of_generations,
        const BoundaryCondition boundary_condition,
        Board_t initial_conditions)
{
    const dim3 dimBlock(32,32);
    const int gridWidth = ceil( ((float)board_width)/32 );
    const dim3 dimGrid(gridWidth, gridWidth);
      
    /* TODO: refer to CUDA C++ Programming Guide, section 3.2.4
     * specifically their implementation of the matrix struct
     * and its associated __device__ functions
     */
}

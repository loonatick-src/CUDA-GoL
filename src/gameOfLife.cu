#include "hip/hip_runtime.h"
#include "gameOfLife.h"

void
gameOfLifeBaseline(const int board_width,
        const int number_of_generations,
        const BoundaryCondition boundary_condition,
        const InitialCondition initial_condition)
{
    const dim3 dimBlock(32,32);
    const int gridWidth = ceil( ((float)board_width)/32 );
    const dim3 dimGrid(gridWidth, gridWidth);

    switch(initial_condition) {
        case (InitialCondition::RANDOM):
            // initialize board randomly
            break;
        case (InitialCondition::CUSTOM):
            // initial board user input
            break;
    }
}
